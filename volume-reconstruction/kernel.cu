#include "hip/hip_runtime.h"
﻿
#include <stdio.h>
#include <string>
#include <vector>
#include <iostream>
#include <chrono>

#include "kernel.hh"

/*
*
*
* 
//*/
//__global__ void delayAndSum(const float* rfData,const float* locData, const GpuConstants* constants, const float* xRange, const float* yRange, const float *zRange, float* volume)
//{
//    // xyz dims 201, 201, 134 
//    int tx = threadIdx.x + blockIdx.x * 8;
//    int ty = threadIdx.y + blockIdx.y * 8;
//    int tz = threadIdx.z + blockIdx.z * 8;
//
//    const float Speed = 1.540f;
//    const float Fs = 100000.0f; // 100 MHz
//    
//    if (tx >= 201 || ty >= 201 || tz >= 134)
//    {
//        return;
//    }
//
//    int voxelId = tz * constants->xCount * constants->yCount + ty * constants->xCount + tx;
//
//    float voxel = volume[ voxelId ];
//    float xPos = xRange[tx];
//    float yPos = yRange[ty];
//    float zPos = zRange[tz];
//
//    float distance;
//    int scanIndex;
//    float exPos, eyPos, ezPos;
//    for (int t = 0; t < constants->transmissionCount; t++)
//    {
//        for (int e = 0; e < constants->elementCount; e++)
//        {
//            exPos = locData[3 * (t * constants->elementCount + e)];
//            eyPos = locData[3 * (t * constants->elementCount + e) + 1];
//            ezPos = locData[3 * (t * constants->elementCount + e) + 2];
//
//            // voxel to rx element
//            distance = sqrtf(powf(xPos - exPos, 2) + powf(yPos - eyPos, 2) + powf(zPos - ezPos, 2)) + zPos;
//
//            // tx element to voxel (only valid for plane waves under the shadow)
//           // distance = distance + zPos;
//
//            scanIndex = (int)roundf(distance * Fs/Speed);
//
//            if (scanIndex >= (constants->rfSampleCount * constants->elementCount * constants->transmissionCount ))
//            {
//                continue;
//            }
//
//            voxel = voxel + rfData[t * constants->rfSampleCount * constants->elementCount + e * constants->rfSampleCount + scanIndex];
//        }
//    }
//
//    volume[voxelId] = voxel;
//
//}

__global__ void delayAndSumFast(const float* rfData, const float* locData, const float* xRange, const float* yRange, const float* zRange, float* volume, int sampleCount, int transmissionCount)
{
    const int elementCount = 508;
    __shared__ float temp[elementCount];
    
    int e = threadIdx.x;

    int2 test;

    if (e >= elementCount)
    {
        return;
    }

    const float samplesPerMeter = 64935.0f; // Fs/c

    const float3 voxPos = { xRange[blockIdx.x], yRange[blockIdx.y], zRange[blockIdx.z] };
    
    float distance;
    int scanIndex;
    float exPos, eyPos;
    for (int t = 0; t < transmissionCount; t++)
    {
        exPos = locData[2 * (t + e * transmissionCount)];
        eyPos = locData[2 * (t + e * transmissionCount) + 1];

        // voxel to rx element
        distance = norm3df(voxPos.x - exPos, voxPos.y - eyPos, voxPos.z) + voxPos.z;

        // tx element to voxel (only valid for plane waves under the shadow)
        // distance = distance + zPos;

        scanIndex = (int)floorf(distance * samplesPerMeter);

        temp[e] += rfData[t * sampleCount * elementCount + e * sampleCount + scanIndex];

    }

    __syncthreads();
    
    // Sum reduction
    int index = 0;
    for (int s = 1; s < elementCount; s *= 2)
    {
        index = 2 * s * e;

        if (index < (elementCount - s))
        {
            temp[index] += temp[index + s];
        }
        
        __syncthreads();
    }

    if (e == 0)
    {
        volume[blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x] = temp[0];
    }
    
}



void
cleanupMemory(float* floats[6])
{
    for (int i = 0; i < 6; i++)
    {
        hipFree(floats[i]);
    }
}

hipError_t volumeReconstruction(Volume* volume, const md::TypedArray<float>& rfData, const md::TypedArray<float>& locData)
{
    float* dRfData = 0;
    float* dLocData = 0;
    float* dVolume = 0;
    
    float* dXPositions = 0;
    float* dYPositions = 0;
    float* dZPositions = 0;

    hipError_t cudaStatus;

    float* deviceData[6] = { dRfData, dLocData, dVolume, dXPositions, dYPositions, dZPositions };

    std::vector<size_t> rfDims = rfData.getDimensions();

    int sampleCount = rfDims[0];
    int transmissionCount = rfDims[2];

    int count;
    hipGetDeviceCount(&count);
    std::cout << count << std::endl;

    // Transfer data to device
    {
        std::cout << "Allocating GPU memory" << std::endl;
        // Choose which GPU to run on, change this on a multi-GPU system.
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to connect to GPU\n");
            cleanupMemory(deviceData);
        }

        // Malloc arrays on GPU
        cudaStatus = hipMalloc((void**)&dRfData, rfData.getNumberOfElements() * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to allocate rf array on device\n");
            cleanupMemory(deviceData);
        }

        cudaStatus = hipMalloc((void**)&dLocData, locData.getNumberOfElements() * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to allocate location array on device\n");
            cleanupMemory(deviceData);
        }

        cudaStatus = hipMalloc((void**)&dVolume, volume->getCount() * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to allocate volume on device\n");
            cleanupMemory(deviceData);
        }

        cudaStatus = hipMalloc((void**)&dXPositions, volume->getXCount() * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to allocate volume on device\n");
            cleanupMemory(deviceData);
        }

        cudaStatus = hipMalloc((void**)&dYPositions, volume->getYCount() * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to allocate volume on device\n");
            cleanupMemory(deviceData);
        }

        cudaStatus = hipMalloc((void**)&dZPositions, volume->getZCount() * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to allocate volume on device\n");
            cleanupMemory(deviceData);
        }

        std::cout << "Transferring data to GPU" << std::endl;

        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(dRfData, (void*)&rfData.begin()[0], rfData.getNumberOfElements() * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to copy RF data to device\n");
            cleanupMemory(deviceData);
        }

        cudaStatus = hipMemcpy(dLocData, (void*)&locData.begin()[0], locData.getNumberOfElements() * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to copy location data to device\n");
            cleanupMemory(deviceData);
        }

        cudaStatus = hipMemcpy(dXPositions, volume->getXRange(), volume->getXCount() * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to copy constants to device\n");
            cleanupMemory(deviceData);
        }

        cudaStatus = hipMemcpy(dYPositions, volume->getYRange(), volume->getYCount() * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to copy constants to device\n");
            cleanupMemory(deviceData);
        }

        cudaStatus = hipMemcpy(dZPositions, volume->getZRange(), volume->getZCount() * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to copy constants to device\n");
            cleanupMemory(deviceData);
        }
    }

    dim3 blockDim(8, 8, 8);
    dim3 gridDim(26, 26, 17);
    std::cout << "Starting kernel" << std::endl;

    auto start = std::chrono::high_resolution_clock::now();

    //delayAndSum<<<gridDim,blockDim>>>(dRfData, dLocData, dConstants, dXPositions, dYPositions, dZPositions, dVolume);

    dim3 gridDim2(201, 201, 134);
    delayAndSumFast<< <gridDim2, 512 >> > (dRfData, dLocData, dXPositions, dYPositions, dZPositions, dVolume, sampleCount, transmissionCount);
    {
        // Transfer Data back
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            cleanupMemory(deviceData);
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = end - start;

        // Print the elapsed time
        std::cout << "Kernel duration: " << elapsed.count() << " seconds" << std::endl;
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after calling kernel\n", cudaStatus);
            cleanupMemory(deviceData);
        }


        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(volume->getData(), dVolume, volume->getCount() * sizeof(float), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to copy volume data out of device\n");
            cleanupMemory(deviceData);
        }
    }


    cleanupMemory(deviceData);

    return cudaStatus;
}


