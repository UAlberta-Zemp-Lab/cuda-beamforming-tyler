#include "hip/hip_runtime.h"
﻿
#include <stdio.h>
#include <string>
#include <vector>
#include <iostream>
#include <chrono>

#include "kernel.hh"

struct GpuConstants {
    const size_t xCount;
    const size_t yCount;
    const size_t zCount;
    const size_t voxelCount;
    const size_t rfSampleCount;
    const size_t elementCount;
    const size_t transmissionCount;
};

/*
*
*
* 
*/
__global__ void delayAndSum(const float* rfData,const float* locData, const GpuConstants* constants, const float* xRange, const float* yRange, const float *zRange, float* volume)
{
    // xyz dims 201, 201, 134 
    int tx = threadIdx.x + blockIdx.x * 8;
    int ty = threadIdx.y + blockIdx.y * 8;
    int tz = threadIdx.z + blockIdx.z * 8;

    const float Speed = 1.540f;
    const float Fs = 100000.0f; // 100 MHz
    
    if (tx >= 201 || ty >= 201 || tz >= 134)
    {
        return;
    }

    int voxelId = tz * constants->xCount * constants->yCount + ty * constants->xCount + tx;

    float voxel = volume[ voxelId ];
    float xPos = xRange[tx];
    float yPos = yRange[ty];
    float zPos = zRange[tz];

    float distance;
    int scanIndex;
    float exPos, eyPos, ezPos;
    for (int t = 0; t < constants->transmissionCount; t++)
    {
        for (int e = 0; e < constants->elementCount; e++)
        {
            exPos = locData[3 * (t * constants->elementCount + e)];
            eyPos = locData[3 * (t * constants->elementCount + e) + 1];
            ezPos = locData[3 * (t * constants->elementCount + e) + 2];

            // voxel to rx element
            distance = sqrtf(powf(xPos - exPos, 2) + powf(yPos - eyPos, 2) + powf(zPos - ezPos, 2)) + zPos;

            // tx element to voxel (only valid for plane waves under the shadow)
           // distance = distance + zPos;

            scanIndex = (int)roundf(distance * Fs/Speed);

            if (scanIndex >= (constants->rfSampleCount * constants->elementCount * constants->transmissionCount ))
            {
                continue;
            }

            voxel = voxel + rfData[t * constants->rfSampleCount * constants->elementCount + e * constants->rfSampleCount + scanIndex];
        }
    }

    volume[voxelId] = voxel;

}

__global__ void delayAndSumFast(const float* rfData, const float* locData, const GpuConstants* constants, const float* xRange, const float* yRange, const float* zRange, float* volume)
{
    __shared__ float temp[508 * 4];

    int tempSize = 508;
    
    int e = threadIdx.x;

    if (e >= 508)
    {
        return;
    }

    int x = blockIdx.x;
    int y = blockIdx.y;
    int z = blockIdx.z;


    const float Speed = 1.540f;
    const float Fs = 100000.0f; // 100 MHz

    float xPos = xRange[x];
    float yPos = yRange[y];
    float zPos = zRange[z];

    float distance;
    int scanIndex;
    float exPos, eyPos, ezPos;

    for (int t = 0; t < constants->transmissionCount; t++)
    {
        exPos = locData[3 * (t * constants->elementCount + e)];
        eyPos = locData[3 * (t * constants->elementCount + e) + 1];
        ezPos = locData[3 * (t * constants->elementCount + e) + 2];

        // voxel to rx element
        distance = sqrtf(powf(xPos - exPos, 2) + powf(yPos - eyPos, 2) + powf(zPos - ezPos, 2)) + zPos;

        // tx element to voxel (only valid for plane waves under the shadow)
        // distance = distance + zPos;

        scanIndex = (int)floorf(distance * Fs / Speed);

        temp[e] += rfData[t * constants->rfSampleCount * constants->elementCount + e * constants->rfSampleCount + scanIndex];


        //voxel = voxel + scan;
    }

    __syncthreads();
    
    for (int s = 1; s < tempSize; s *= 2)
    {
        int index = 2 * s * e;

        if (index < (tempSize - s))
        {
            temp[index] += temp[index + s];
        }
        
        __syncthreads();
    }

    if (e == 0)
    {
        volume[z * constants->xCount * constants->yCount + y * constants->xCount + x] = temp[0];
    }
    
}



void
cleanupMemory(float* floats[6], GpuConstants* constants)
{
    hipFree(constants);

    for (int i = 0; i < 6; i++)
    {
        hipFree(floats[i]);
    }
}

hipError_t volumeReconstruction(Volume* volume, const md::TypedArray<float>& rfData, const md::TypedArray<float>& locData)
{
    float* dRfData = 0;
    float* dLocData = 0;
    float* dVolume = 0;
    GpuConstants* dConstants = 0;
    hipError_t cudaStatus;
    
    float* dXPositions = 0;
    float* dYPositions = 0;
    float* dZPositions = 0;

    float* deviceData[6] = { dRfData, dLocData, dVolume, dXPositions, dYPositions, dZPositions };

    std::vector<size_t> rfDims = rfData.getDimensions();

    GpuConstants constants = {
        volume->getXCount(),
        volume->getYCount(),
        volume->getZCount(),
        volume->getCount(),
        rfDims[0],
        rfDims[1],
        rfDims[2] };

    int count;
    hipGetDeviceCount(&count);
    std::cout << count << std::endl;

    // Transfer data to device
    {
        std::cout << "Allocating GPU memory" << std::endl;
        // Choose which GPU to run on, change this on a multi-GPU system.
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to connect to GPU\n");
            cleanupMemory(deviceData, dConstants);
        }

        // Malloc arrays on GPU
        cudaStatus = hipMalloc((void**)&dRfData, rfData.getNumberOfElements() * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to allocate rf array on device\n");
            cleanupMemory(deviceData, dConstants);
        }

        cudaStatus = hipMalloc((void**)&dLocData, locData.getNumberOfElements() * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to allocate location array on device\n");
            cleanupMemory(deviceData, dConstants);
        }

        cudaStatus = hipMalloc((void**)&dConstants, sizeof(GpuConstants));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to allocate constants array on device\n");
            cleanupMemory(deviceData, dConstants);
        }

        cudaStatus = hipMalloc((void**)&dVolume, volume->getCount() * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to allocate volume on device\n");
            cleanupMemory(deviceData, dConstants);
        }

        cudaStatus = hipMalloc((void**)&dXPositions, volume->getXCount() * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to allocate volume on device\n");
            cleanupMemory(deviceData, dConstants);
        }

        cudaStatus = hipMalloc((void**)&dYPositions, volume->getYCount() * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to allocate volume on device\n");
            cleanupMemory(deviceData, dConstants);
        }

        cudaStatus = hipMalloc((void**)&dZPositions, volume->getZCount() * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to allocate volume on device\n");
            cleanupMemory(deviceData, dConstants);
        }

        std::cout << "Transferring data to GPU" << std::endl;

        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(dRfData, (void*)&rfData.begin()[0], rfData.getNumberOfElements() * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to copy RF data to device\n");
            cleanupMemory(deviceData, dConstants);
        }

        cudaStatus = hipMemcpy(dLocData, (void*)&locData.begin()[0], locData.getNumberOfElements() * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to copy location data to device\n");
            cleanupMemory(deviceData, dConstants);
        }

        cudaStatus = hipMemcpy(dConstants, &constants, sizeof(GpuConstants), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to copy constants to device\n");
            cleanupMemory(deviceData, dConstants);
        }

        cudaStatus = hipMemcpy(dXPositions, volume->getXRange(), volume->getXCount() * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to copy constants to device\n");
            cleanupMemory(deviceData, dConstants);
        }

        cudaStatus = hipMemcpy(dYPositions, volume->getYRange(), volume->getYCount() * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to copy constants to device\n");
            cleanupMemory(deviceData, dConstants);
        }

        cudaStatus = hipMemcpy(dZPositions, volume->getZRange(), volume->getZCount() * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to copy constants to device\n");
            cleanupMemory(deviceData, dConstants);
        }
    }

    dim3 blockDim(8, 8, 8);
    dim3 gridDim(26, 26, 17);
    std::cout << "Starting kernel" << std::endl;

    auto start = std::chrono::high_resolution_clock::now();

    delayAndSum<<<gridDim,blockDim>>>(dRfData, dLocData, dConstants, dXPositions, dYPositions, dZPositions, dVolume);

    dim3 blockDim2(201, 201, 134);
    delayAndSumFast<< <blockDim2, 512 >> > (dRfData, dLocData, dConstants, dXPositions, dYPositions, dZPositions, dVolume);
    {
        // Transfer Data back
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            cleanupMemory(deviceData, dConstants);
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = end - start;

        // Print the elapsed time
        std::cout << "Kernel duration: " << elapsed.count() << " seconds" << std::endl;
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after calling kernel\n", cudaStatus);
            cleanupMemory(deviceData, dConstants);
        }


        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(volume->getData(), dVolume, volume->getCount() * sizeof(float), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Failed to copy volume data out of device\n");
            cleanupMemory(deviceData, dConstants);
        }
    }


    cleanupMemory(deviceData, dConstants);

    return cudaStatus;
}


