#include "hip/hip_runtime.h"
﻿


#include <stdio.h>
#include <string>
#include <vector>
#include <iostream>

#include "kernel.hh"

struct GpuConstants {
    const int xCount;
    const int yCount;
    const int zCount;
    const int voxelCount;
    const int transmissionCount;
    const int elementCount;
    const int rfSampleCount;
};


__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}


__global__ void delayAndSum(const float* rfData,const float* locData, const GpuConstants* constants, const float* xRange, const float* yRange, const float *zRange, float* volume)
{
    // xyz dims 201, 201, 134 
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;

    const float Speed = 1540;
    const float Fs = 100000000; // 100 MHz
    
    if (tx != 101 || ty >= 201 || tz >= 134)
    {
        return;
    }

    int voxelId = (tx * constants->yCount * constants->zCount) + (ty * constants->zCount) + tz;

    float* voxel = &volume[ voxelId ];
    float xPos = xRange[tx];
    float yPos = yRange[ty];
    float zPos = zRange[tz];

    float distance;
    int scanIndex;
    float exPos, eyPos, ezPos;
    for (int t = 0; t < constants->transmissionCount; t++)
    {
        for (int e = 0; e < constants->elementCount; e++)
        {
            exPos = locData[3 * (t * constants->elementCount + e)];
            eyPos = locData[3 * (t * constants->elementCount + e) + 1];
            ezPos = locData[3 * (t * constants->elementCount + e) + 2];

            // voxel to rx element
            distance = sqrtf(powf(xPos - exPos, 2) + powf(yPos - eyPos, 2) + powf(zPos - ezPos, 2));

            // tx element to voxel (only valid for plane waves under the shadow)
            distance = distance + zPos;

            scanIndex = roundf(distance / (Speed * Fs));

            if (scanIndex >= constants->rfSampleCount)
            {
                continue;
            }

            *voxel = *voxel + rfData[t * constants->rfSampleCount + scanIndex];
        }
    }

}

hipError_t volumeReconstruction(Volume* volume, const CellDataArray& rfData, const CellDataArray& locData)
{
    float* dRfData = 0;
    float* dLocData = 0;
    float* dVolume = 0;
    GpuConstants* dConstants = 0;
    hipError_t cudaStatus;
    
    float* dXPositions = 0;
    float* dYPositions = 0;
    float* dZPositions = 0;


    GpuConstants constants = {
        volume->getXCount(),
        volume->getYCount(),
        volume->getZCount(),
        volume->getCount(),
        rfData.getCellCount(),
        rfData.getColumnCount(),
        rfData.getRowCount() };

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to connect GPU\n");
        goto Error;
    }

    // Malloc arrays on GPU
    cudaStatus = hipMalloc((void**)&dRfData, rfData.getCount() * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to allocate rf array on device\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dLocData, locData.getCount() * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to allocate location array on device\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dConstants, sizeof(GpuConstants));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to allocate constants array on device\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dVolume, volume->getCount() * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to allocate volume on device\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dXPositions, volume->getXCount() * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to allocate volume on device\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dYPositions, volume->getYCount() * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to allocate volume on device\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dZPositions, volume->getZCount() * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to allocate volume on device\n");
        goto Error;
    }



    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dRfData, rfData.getData(), rfData.getCount() * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to copy RF data to device\n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dLocData, locData.getData(), locData.getCount() * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to copy location data to device\n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dConstants, &constants, sizeof(GpuConstants), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to copy constants to device\n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dXPositions, volume->_xRange.data(), 201 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to copy constants to device\n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dYPositions, volume->_yRange.data(), 201 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to copy constants to device\n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dZPositions, volume->_xRange.data(), 134 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to copy constants to device\n");
        goto Error;
    }

    // KERNEL


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after calling kernel\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(volume->getData(), dVolume, volume->getCount() * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to copy volume data out of device\n");
        goto Error;
    }


Error:
    hipFree(dRfData);
    hipFree(dLocData);
    hipFree(dConstants);
    hipFree(dVolume);
    hipFree(dXPositions);
    hipFree(dYPositions);
    hipFree(dZPositions);

    return cudaStatus;
}